/*************************************************************************** *cr *cr (C) Copyright 2010 The Board of Trustees of the
 *cr University of Illinois *cr All Rights Reserved *cr ***************************************************************************/ /* * Main entry of dense matrix-matrix multiplication kernel */ #include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <malloc.h>
#include <vector>
#include <iostream>
#include "sgemm_kernel.cu"
 extern bool readColMajorMatrixFile(const char *fn, int &nr_row, int &nr_col, std::vector<float>&v);
extern bool writeColMajorMatrixFile(const char *fn, int, int, std::vector<float>&);
 extern "C" void computeGold(float *, const float*, const float*, unsigned int, unsigned int, unsigned int);
 int main (int argc, char *argv[]) {
 float *dA, *dB, *dC;
 size_t A_sz, B_sz, C_sz;
 int matArow, matAcol;
 int matBrow, matBcol;
 std::vector<float> matA, matBT;
 int n;
 if(argc == 2){
 n = atoi(argv[1]);
A_sz = n * n * sizeof(float);
B_sz = A_sz;
C_sz = A_sz;
matArow = n;
matAcol = n;
matBrow = n;
matBcol = n;
 }else{
printf("error in input");
return 0;
 } 
 for(int i = 0; i < n; i++)
 for(int j = 0; j < n; j++){
 matA.push_back(static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/40))); 
 matBT.push_back(static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/40)));
 }
 std::vector<float> matC(matArow*matBcol);
 hipMalloc((void**)&dA, A_sz);
 hipMalloc((void**)&dB, B_sz);
 hipMalloc((void**)&dC, C_sz);
 hipMemcpy(dA, &matA.front(), A_sz, hipMemcpyHostToDevice); 
 hipMemcpy(dB, &matBT.front(), B_sz, hipMemcpyHostToDevice); 
 regtileSgemm('N', 'T', matArow, matBcol, matAcol, 1.0f, dA, matArow, dB, matBcol, 0.0f, dC, matArow);
 hipMemcpy(&matC.front(), dC, C_sz, hipMemcpyDeviceToHost);
 for(int i = 100; i < 103; i++){
 for(int j = 100; j < 103; j++)
 printf("%d ", matC.at(i * n + j));
printf("\n");
 }
 hipFree(dA);
 hipFree(dB);
 hipFree(dC);
 return 0;
}
