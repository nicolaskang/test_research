#include "hip/hip_runtime.h"
 #include "monteCarloKernels.cuh"
 #define A_1 -39.696830286653757
#define A_2 220.94609842452050
#define A_3 -275.92851044696869
#define A_4 138.35775186726900
#define A_5 -30.664798066147160
#define A_6 2.5066282774592392
#define B_1 -54.476098798224058
#define B_2 161.58583685804089
#define B_3 -155.69897985988661
#define B_4 66.801311887719720
#define B_5 -13.280681552885721
 __global__ void setup_kernel ( hiprandState * state, int seedVal, int numSamples )
{
int id = blockIdx.x*blockDim.x + threadIdx.x;
if (id < numSamples)
{
/* Each thread gets same seed , a different sequence number , no offset */ hiprand_init (seedVal , id , 0 , &(state[id])) ;
}
}
 __device__ dataType compInverseNormDist(dataType x) 
{
dataType z;
 z = x - 0.5;
 dataType r = z*z;
 z = (((((A_1*r+A_2)*r+A_3)*r+A_4)*r+A_5)*r+A_6)*z /
 (((((B_1*r+B_2)*r+B_3)*r+B_4)*r+B_5)*r+1.0);
return z;
}
 __device__ dataType interestRateCompoundFact(dataType t, dataType rate)
{
 return exp(rate*t);
}
 __device__ dataType interestRateDiscountFact(dataType t, dataType rate)
{
return 1.0/interestRateCompoundFact(t, rate);
}
 __device__ dataType flatForwardDiscountImpl(dataType t, dataType rate)
{
return interestRateDiscountFact(t, rate);
}
 __device__ dataType yieldTermStructDiscount(dataType t, dataType rate)
{
return flatForwardDiscountImpl(t, rate);
}
 __device__ dataType interestRateImpliedRate(dataType compound, dataType t)
{
dataType r = log(compound) / t;
return r;
}
 __device__ dataType yieldTermStructForwardRate(dataType t1, dataType t2, dataType rate)
{
dataType compound = interestRateDiscountFact(t1, rate) / interestRateDiscountFact(t2, rate);
return interestRateImpliedRate(compound, t2 - t1);
}
 __device__ dataType processDrift(dataType t, dataType x, monteCarloOptionStruct optionStruct)
{
dataType sigma = processDiff(t, x, optionStruct);
dataType t1 = t + 0.0001;
 return yieldTermStructForwardRate(t, t1, optionStruct.riskVal) - yieldTermStructForwardRate(t, t1, optionStruct.divVal) - (0.5 * sigma * sigma);
} 
 __device__ dataType discretizationDrift(dataType t0, dataType x0, dataType dt, monteCarloOptionStruct optionStruct)
{
return processDrift(t0, x0, optionStruct) * dt;
}
 __device__ dataType localVoltLocVol(dataType t, dataType underlyingLevel, monteCarloOptionStruct optionStruct)
{
return optionStruct.voltVal;
}
 __device__ dataType processDiff(dataType t, dataType x, monteCarloOptionStruct optionStruct)
{
return localVoltLocVol(t, x, optionStruct);
}
 __device__ dataType discDiff(dataType t0, dataType x0, dataType dt, monteCarloOptionStruct optionStruct)
{
return processDiff(t0, x0, optionStruct) * sqrt(dt);
}
 __device__ dataType stdDeviation(dataType t0, dataType x0, dataType dt, monteCarloOptionStruct optionStruct)
{
return discDiff(t0, x0, dt, optionStruct);
}
 __device__ dataType apply(dataType x0, dataType dx)
{
return (x0 * exp(dx));
}
 __device__ dataType discDrift(dataType t0, dataType x0, dataType dt, monteCarloOptionStruct optionStruct)
{
return processDrift(t0, x0, optionStruct) * dt;
}
 __device__ dataType processEvolve(dataType t0, dataType x0, dataType dt, dataType dw, monteCarloOptionStruct optionStruct)
{
return apply(x0, discDrift(t0, x0, dt, optionStruct) + stdDeviation(t0, x0, dt, optionStruct) *dw);
}
 __device__ void getSequence(dataType* sequence, dataType sampleNum)
{
for (size_t iInSeq = 0; iInSeq < SEQUENCE_LENGTH; iInSeq++)
{
sequence[iInSeq] = DEFAULT_SEQ_VAL;
}
}
 __device__ dataType getProcessValX0(monteCarloOptionStruct optionStruct)
{
return optionStruct.underlyingVal;
}
 __device__ void getPath(dataType* path, size_t sampleNum, dataType dt, hiprandState * state, monteCarloOptionStruct optionStruct)
{
 path[0] = getProcessValX0(optionStruct);
 for (size_t i=1; i<SEQUENCE_LENGTH; i++) 
{
dataType t = i*dt; 
dataType randVal = hiprand_uniform (& (state[sampleNum]) ) ;
dataType inverseCumRandVal = compInverseNormDist(randVal); 
path[i] = processEvolve(t, path[i-1], dt, inverseCumRandVal, optionStruct); 
}
}
 __device__ dataType getPrice(dataType val)
{
return max(STRIKE_VAL-val, 0.0)* DISCOUNT_VAL;
}
 __device__ void initializePath(dataType* path)
{
for (int i=0; i < SEQUENCE_LENGTH; i++)
{
path[i] = START_PATH_VAL;
}
}
 __global__ void monteCarloGpuKernel(dataType* samplePrices, dataType* sampleWeights, dataType* times, dataType dt, hiprandState * state, monteCarloOptionStruct* optionStructs, int numSamples)
{
 size_t numThread = blockIdx.x*blockDim.x + threadIdx.x;
 int numOption = 0;
 int numSample = numThread;
 size_t outputNum = numSample;
 if (numSample < numSamples)
{
 dataType path[SEQUENCE_LENGTH];
initializePath(path);
 getPath(path, numSample, dt, state, optionStructs[numOption]);
dataType price = getPrice(path[SEQUENCE_LENGTH-1]);
 samplePrices[outputNum] = price;
sampleWeights[outputNum] = DEFAULT_SEQ_WEIGHT;
 }
}
