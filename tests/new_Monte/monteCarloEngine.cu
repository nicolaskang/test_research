#include "hip/hip_runtime.h"
 #include "monteCarloKernels.cu"
 #include "monteCarloKernelsCpu.cu"
 #include <hiprand/hiprand_kernel.h>
 /* #include <thrust/transform_reduce.h> */
/* #include <thrust/functional.h> */
/* #include <thrust/device_vector.h> */
/* #include <thrust/host_vector.h> */
 #include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
 #define RISK_VAL 0.06f
#define DIV_VAL 0.0f
#define VOLT_VAL 0.200f
#define UNDERLYING_VAL 30.0f
#define STRIKE_VAL 40.0f
#define DISCOUNT_VAL 0.94176453358424872f
 void initializeInputs(dataType* samplePrices, dataType* sampleWeights, dataType* times)
{
}
 void runMonteCarlo()
{
 int numSamples = 400000;
 {
 printf("Number of Samples: %d\n\n", numSamples);
 monteCarloOptionStruct optionStruct;
optionStruct.riskVal = RISK_VAL;
optionStruct.divVal = DIV_VAL;
optionStruct.voltVal = VOLT_VAL;
optionStruct.underlyingVal = UNDERLYING_VAL;
optionStruct.strikeVal = STRIKE_VAL;
optionStruct.discountVal = DISCOUNT_VAL;
 dataType* samplePrices;
dataType* sampleWeights;
dataType* times;
monteCarloOptionStruct* optionStructs;
 samplePrices = (dataType*)malloc(NUM_OPTIONS*numSamples*sizeof(dataType));
sampleWeights = (dataType*)malloc(NUM_OPTIONS*numSamples*sizeof(dataType));
times = (dataType*)malloc(NUM_OPTIONS*numSamples*sizeof(dataType));
optionStructs = (monteCarloOptionStruct*)malloc(NUM_OPTIONS*sizeof(monteCarloOptionStruct));
 long seconds, useconds;
dataType mtimeGpu, mtimeCpu; 
struct timeval start;
struct timeval end;
 for (int optNum = 0; optNum < NUM_OPTIONS; optNum++)
{
optionStructs[optNum] = optionStruct;
}
 dataType* samplePricesGpu;
dataType* sampleWeightsGpu;
dataType* timesGpu;
monteCarloOptionStruct* optionStructsGpu;
 hiprandState* devStates;
 /* Allocate space for prng states on device */ hipMalloc (( void **) & devStates , numSamples * sizeof ( hiprandState ) );
 hipMalloc(&samplePricesGpu, NUM_OPTIONS*numSamples*sizeof(dataType));
hipMalloc(&sampleWeightsGpu, NUM_OPTIONS*numSamples*sizeof(dataType));
hipMalloc(&timesGpu, NUM_OPTIONS*numSamples*sizeof(dataType));
hipMalloc(&optionStructsGpu, NUM_OPTIONS*sizeof(monteCarloOptionStruct));
 hipMemcpy(samplePricesGpu, samplePrices, NUM_OPTIONS*numSamples*sizeof(dataType), hipMemcpyHostToDevice);
hipMemcpy(sampleWeightsGpu, sampleWeights, NUM_OPTIONS*numSamples*sizeof(dataType), hipMemcpyHostToDevice);
hipMemcpy(timesGpu, times, NUM_OPTIONS*numSamples*sizeof(dataType), hipMemcpyHostToDevice);
hipMemcpy(optionStructsGpu, optionStructs, NUM_OPTIONS*sizeof(monteCarloOptionStruct), hipMemcpyHostToDevice);
 srand(time(NULL));
 {
 /* initialize random seed: */ srand ( rand());
 printf("Run on GPU\n");
gettimeofday(&start, NULL);
 dim3 grid1( (size_t)ceil((dataType)numSamples / ((dataType)THREAD_BLOCK_SIZE)), 1, 1);
dim3 threads1( THREAD_BLOCK_SIZE, 1, 1);
 setup_kernel <<< grid1, threads1 >>> (devStates, rand(), numSamples);
 hipDeviceSynchronize();
 dim3 grid2( (size_t)ceil((dataType)numSamples / (THREAD_BLOCK_SIZE)), 1, 1);
 dim3 threads2( THREAD_BLOCK_SIZE, 1, 1);
 monteCarloGpuKernel <<< grid2, threads2 >>>(samplePricesGpu, sampleWeightsGpu, timesGpu, (1.0f / (dataType)SEQUENCE_LENGTH), devStates, optionStructsGpu, numSamples);
 hipDeviceSynchronize();
 gettimeofday(&end, NULL);
 seconds = end.tv_sec - start.tv_sec;
 useconds = end.tv_usec - start.tv_usec;
 mtimeGpu = ((seconds) * 1000 + ((dataType)useconds)/1000.0) + 0.5;
 printf("Processing time on GPU (CUDA): %f (ms)\n", mtimeGpu);
 /* /* thrust::device_ptr<dataType> devicePointerPrices(samplePricesGpu); */
 /* /* for (int numOpt=0; numOpt < NUM_OPTIONS; numOpt++) */
/* { */
/* /* dataType sum = thrust::reduce(devicePointerPrices + numOpt*numSamples, devicePointerPrices + (numOpt+1)*(numSamples), (dataType) 0, thrust::plus<dataType>()); */
 /* dataType avgPrice = sum / numSamples; */
/* printf("avgPrice: %f\n", avgPrice); */
/* } */
 hipMemcpy(samplePrices, samplePricesGpu, numSamples*sizeof(dataType), hipMemcpyDeviceToHost);
 hipMemcpy(sampleWeights, sampleWeightsGpu, numSamples*sizeof(dataType), hipMemcpyDeviceToHost);
hipMemcpy(times, timesGpu, numSamples*sizeof(dataType), hipMemcpyDeviceToHost);
 dataType cumPrice = 0.0f;
 for (int numSamp = 0; numSamp < numSamples; numSamp++)
{
cumPrice += samplePrices[numSamp];
}
 dataType avgPrice = cumPrice / numSamples;
printf("Average Price (GPU computation): %f\n\n", avgPrice);
 }
 hipFree(samplePricesGpu);
hipFree(sampleWeightsGpu);
hipFree(timesGpu);
hipFree(optionStructsGpu);
 free(samplePrices);
free(sampleWeights);
free(times);
 dataType* samplePricesCpu;
dataType* sampleWeightsCpu;
dataType* timesCpu;
 samplePricesCpu = (dataType*)malloc(numSamples*sizeof(dataType));
sampleWeightsCpu = (dataType*)malloc(numSamples*sizeof(dataType));
timesCpu = (dataType*)malloc(numSamples*sizeof(dataType));
 gettimeofday(&start, NULL);
 monteCarloGpuKernelCpu(samplePricesCpu, sampleWeightsCpu, timesCpu, (1.0f / (dataType)SEQUENCE_LENGTH), optionStructs, numSamples);
 gettimeofday(&end, NULL);
 seconds = end.tv_sec - start.tv_sec;
useconds = end.tv_usec - start.tv_usec;
 mtimeCpu = ((seconds) * 1000 + ((dataType)useconds)/1000.0) + 0.5;
printf("Run on CPU\n");
printf("Processing time on CPU: %f (ms)\n", mtimeCpu);
 dataType cumPrice = 0.0f;
 for (int numSamp = 0; numSamp < numSamples; numSamp++)
{
cumPrice += samplePricesCpu[numSamp];
}
 dataType avgPrice = cumPrice / numSamples;
printf("Average Price (CPU computation): %f\n\n", avgPrice);
 printf("Speedup on GPU: %f\n", mtimeCpu / mtimeGpu);
 free(samplePricesCpu);
free(sampleWeightsCpu);
free(timesCpu);
free(optionStructs);
 }
}
 int main( int argc, char** argv) 
{
runMonteCarlo();
 char c;
c = getchar();
printf("%c\n", c);
}
