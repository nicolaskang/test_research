#include "hip/hip_runtime.h"
#include "test.cu"
//blackScholesAnalyticEngineKernels.cu
//Scott Grauer-Gray
//Kernels for running black scholes using the analytic engine

#ifndef BLACK_SCHOLES_ANALYTIC_ENGINE_KERNELS_CUH
#define BLACK_SCHOLES_ANALYTIC_ENGINE_KERNELS_CUH

//declarations for the kernels
#include "blackScholesAnalyticEngineKernels.cuh"

//needed for the constants in the error function
#include "errorFunctConsts.cuh"


//device kernel to retrieve the compound factor in interestRate
__device__ float interestRateCompoundFactor(float t, yieldTermStruct currYieldTermStruct, vals* localvars )
{
	return (exp((currYieldTermStruct.forward)*t));
}


//device kernel to retrieve the discount factor in interestRate
__device__ float interestRateDiscountFactor(float t, yieldTermStruct currYieldTermStruct, vals* localvars )
{
	return 1.0f / interestRateCompoundFactor(t, currYieldTermStruct,localvars);
}


//device function to get the variance of the black volatility function
__device__ float getBlackVolBlackVar(blackVolStruct volTS, vals* localvars )
{
	float vol = volTS.volatility;
	localvars[CR_threadIndex].vol=vol;
	return vol*vol*volTS.timeYearFraction;
}


//device function to get the discount on a dividend yield
__device__ float getDiscountOnDividendYield(float yearFraction, yieldTermStruct dividendYieldTermStruct, vals* localvars )
{
	float intDiscountFactor = interestRateDiscountFactor(yearFraction, dividendYieldTermStruct,localvars);
	localvars[CR_threadIndex].intDiscountFactor=intDiscountFactor;
	return intDiscountFactor;
}


//device function to get the discount on the risk free rate
__device__ float getDiscountOnRiskFreeRate(float yearFraction, yieldTermStruct riskFreeRateYieldTermStruct, vals* localvars )
{
	return interestRateDiscountFactor(yearFraction, riskFreeRateYieldTermStruct,localvars);
}


//device kernel to run the error function
__device__ float errorFunct(normalDistStruct normDist, float x, vals* localvars )
{
	float R,S,P,Q,s,y,z,r, ax;

    ax = fabs(x);
	localvars[CR_threadIndex].ax=ax;

    if(ax < 0.84375) 
	{      
        if(ax < 3.7252902984e-09) 
		{ 
                if (ax < DBL_MIN*16)
                    return 0.125*(8.0*x+ (ERROR_FUNCT_efx8)*x);  /*avoid underflow */
	            return x + (ERROR_FUNCT_efx)*x;
        }
		z = x*x;
	localvars[CR_threadIndex].z=z;
        r = ERROR_FUNCT_pp0+z*(ERROR_FUNCT_pp1+z*(ERROR_FUNCT_pp2+z*(ERROR_FUNCT_pp3+z*ERROR_FUNCT_pp4)));
	localvars[CR_threadIndex].r=r;
        s = ERROR_FUNCT_one+z*(ERROR_FUNCT_qq1+z*(ERROR_FUNCT_qq2+z*(ERROR_FUNCT_qq3+z*(ERROR_FUNCT_qq4+z*ERROR_FUNCT_qq5))));
	localvars[CR_threadIndex].s=s;
        y = r/s;
	localvars[CR_threadIndex].y=y;
        return x + x*y;
    }
    if(ax <1.25) 
	{      
        s = ax-ERROR_FUNCT_one;
	localvars[CR_threadIndex].s=s;
        P = ERROR_FUNCT_pa0+s*(ERROR_FUNCT_pa1+s*(ERROR_FUNCT_pa2+s*(ERROR_FUNCT_pa3+s*(ERROR_FUNCT_pa4+s*(ERROR_FUNCT_pa5+s*ERROR_FUNCT_pa6)))));
	localvars[CR_threadIndex].P=P;
        Q = ERROR_FUNCT_one+s*(ERROR_FUNCT_qa1+s*(ERROR_FUNCT_qa2+s*(ERROR_FUNCT_qa3+s*(ERROR_FUNCT_qa4+s*(ERROR_FUNCT_qa5+s*ERROR_FUNCT_qa6)))));
	localvars[CR_threadIndex].Q=Q;
        if(x>=0) return ERROR_FUNCT_erx + P/Q; else return -1*ERROR_FUNCT_erx - P/Q;
    }
    if (ax >= 6) 
	{      
        if(x>=0) 
			return ERROR_FUNCT_one-ERROR_FUNCT_tiny; 
		else 
			return ERROR_FUNCT_tiny-ERROR_FUNCT_one;
    }

    /* Starts to lose accuracy when ax~5 */
    s = ERROR_FUNCT_one/(ax*ax);
	localvars[CR_threadIndex].s=s;

    if(ax < 2.85714285714285) { /* |x| < 1/0.35 */
        R = ERROR_FUNCT_ra0+s*(ERROR_FUNCT_ra1+s*(ERROR_FUNCT_ra2+s*(ERROR_FUNCT_ra3+s*(ERROR_FUNCT_ra4+s*(ERROR_FUNCT_ra5+s*(ERROR_FUNCT_ra6+s*ERROR_FUNCT_ra7))))));
	localvars[CR_threadIndex].R=R;
        S = ERROR_FUNCT_one+s*(ERROR_FUNCT_sa1+s*(ERROR_FUNCT_sa2+s*(ERROR_FUNCT_sa3+s*(ERROR_FUNCT_sa4+s*(ERROR_FUNCT_sa5+s*(ERROR_FUNCT_sa6+s*(ERROR_FUNCT_sa7+s*ERROR_FUNCT_sa8)))))));
	localvars[CR_threadIndex].S=S;
    } else {    /* |x| >= 1/0.35 */
        R=ERROR_FUNCT_rb0+s*(ERROR_FUNCT_rb1+s*(ERROR_FUNCT_rb2+s*(ERROR_FUNCT_rb3+s*(ERROR_FUNCT_rb4+s*(ERROR_FUNCT_rb5+s*ERROR_FUNCT_rb6)))));
	localvars[CR_threadIndex].R=R;
        S=ERROR_FUNCT_one+s*(ERROR_FUNCT_sb1+s*(ERROR_FUNCT_sb2+s*(ERROR_FUNCT_sb3+s*(ERROR_FUNCT_sb4+s*(ERROR_FUNCT_sb5+s*(ERROR_FUNCT_sb6+s*ERROR_FUNCT_sb7))))));
	localvars[CR_threadIndex].S=S;
    }

    r = exp( -ax*ax-0.5625 +R/S);
	localvars[CR_threadIndex].r=r;
    if(x>=0) 
		return ERROR_FUNCT_one-r/ax; 
	else 
		return r/ax-ERROR_FUNCT_one;
}



//device kernel to run the operator function in cumulative normal distribution
__device__ float cumNormDistOp(normalDistStruct normDist, float z, vals* localvars )
{
	z = (z - normDist.average) / normDist.sigma;
	localvars[CR_threadIndex].z=z;
    	float result = 0.5 * ( 1.0 + errorFunct(normDist, z*M_SQRT_2 ,localvars) );
	localvars[CR_threadIndex].result=result;
	return result;
}


//device kernel to run the gaussian function in the normal distribution
__device__ float gaussianFunctNormDist(normalDistStruct normDist, float x, vals* localvars )
{
	float deltax = x - normDist.average;
	localvars[CR_threadIndex].deltax=deltax;
	float exponent = -(deltax*deltax)/normDist.denominator;
	localvars[CR_threadIndex].exponent=exponent;

    	// debian alpha had some strange problem in the very-low range
    	return exponent <= -690.0 ? 0.0 :  // exp(x) < 1.0e-300 anyway
            normDist.normalizationFactor * exp(exponent);
}


//device kernel to retrieve the derivative in a cumulative normal distribution
__device__ float cumNormDistDeriv(normalDistStruct normDist, float x, vals* localvars )
{
	float xn = (x - normDist.average) / normDist.sigma;
	localvars[CR_threadIndex].xn=xn;
    	return gaussianFunctNormDist(normDist, xn,localvars) / normDist.sigma;
}


//device function to initialize the cumulative normal distribution structure
__device__ void initCumNormDist(normalDistStruct& currCumNormDist, vals* localvars )
{
	currCumNormDist.average = 0.0f;
	localvars[CR_threadIndex].currCumNormDist->average=currCumNormDist->average;
	currCumNormDist.sigma = 1.0f;
	localvars[CR_threadIndex].currCumNormDist->sigma=currCumNormDist->sigma;
	currCumNormDist.normalizationFactor = M_SQRT_2*M_1_SQRTPI/currCumNormDist.sigma;
	localvars[CR_threadIndex].currCumNormDist->normalizationFactor=currCumNormDist->normalizationFactor;
    	currCumNormDist.derNormalizationFactor = currCumNormDist.sigma*currCumNormDist.sigma;
	localvars[CR_threadIndex].currCumNormDist->derNormalizationFactor=currCumNormDist->derNormalizationFactor;
    	currCumNormDist.denominator = 2.0*currCumNormDist.derNormalizationFactor;
	localvars[CR_threadIndex].currCumNormDist->denominator=currCumNormDist->denominator;
}


//device function to initialize variable in the black calculator
__device__ void initBlackCalcVars(blackCalcStruct& blackCalculator, payoffStruct payoff, vals* localvars )
{
	blackCalculator.d1 = log(blackCalculator.forward / blackCalculator.strike)/blackCalculator.stdDev + 0.5*blackCalculator.stdDev;
	blackCalculator.d2 = blackCalculator.d1 - blackCalculator.stdDev;

	//initialize the cumulative normal distribution structure
	normalDistStruct currCumNormDist;
	initCumNormDist(currCumNormDist,localvars);
	localvars[CR_threadIndex].currCumNormDist=currCumNormDist;
                
	blackCalculator.cum_d1 = cumNormDistOp(currCumNormDist, blackCalculator.d1,localvars);
	blackCalculator.cum_d2 = cumNormDistOp(currCumNormDist, blackCalculator.d2,localvars);
	blackCalculator.n_d1 = cumNormDistDeriv(currCumNormDist, blackCalculator.d1,localvars);
	blackCalculator.n_d2 = cumNormDistDeriv(currCumNormDist, blackCalculator.d2,localvars);

	blackCalculator.x = payoff.strike;
	blackCalculator.DxDstrike = 1.0;

	// the following one will probably disappear as soon as
	// super-share will be properly handled
	blackCalculator.DxDs = 0.0;

	// this part is always executed.
	// in case of plain-vanilla payoffs, it is also the only part
	// which is executed.
	switch (payoff.type) 
	{
		case CALL:
		    blackCalculator.alpha     =  blackCalculator.cum_d1;//  N(d1)
		    blackCalculator.DalphaDd1 =    blackCalculator.n_d1;//  n(d1)
		    blackCalculator.beta      = -1.0f*blackCalculator.cum_d2;// -N(d2)
		    blackCalculator.DbetaDd2  = -1.0f*blackCalculator.n_d2;// -n(d2)
		    break;
		case PUT:
		    blackCalculator.alpha     = -1.0+blackCalculator.cum_d1;// -N(-d1)
		    blackCalculator.DalphaDd1 =        blackCalculator.n_d1;//  n( d1)
		    blackCalculator.beta      =  1.0-blackCalculator.cum_d2;//  N(-d2)
		    blackCalculator.DbetaDd2  =     -1.0f* blackCalculator.n_d2;// -n( d2)
		    break;
	}
}


//device function to initialize the black calculator
__device__ void initBlackCalculator(blackCalcStruct& blackCalc, payoffStruct payoff, float forwardPrice, float stdDev, float riskFreeDiscount, vals* localvars )
{
	blackCalc.strike = payoff.strike;
	localvars[CR_threadIndex].blackCalc->strike=blackCalc->strike;
	blackCalc.forward = forwardPrice;
	localvars[CR_threadIndex].blackCalc->forward=blackCalc->forward;
	blackCalc.stdDev = stdDev;
	localvars[CR_threadIndex].blackCalc->stdDev=blackCalc->stdDev;
	blackCalc.discount = riskFreeDiscount;
	localvars[CR_threadIndex].blackCalc->discount=blackCalc->discount;
	blackCalc.variance = stdDev * stdDev;
	localvars[CR_threadIndex].blackCalc->variance=blackCalc->variance;

	initBlackCalcVars(blackCalc, payoff,localvars);
	localvars[CR_threadIndex].blackCalc=blackCalc;
}


//device function to retrieve the output resulting value
__device__ float getResultVal(blackCalcStruct blackCalculator, vals* localvars )
{
	float result = blackCalculator.discount * (blackCalculator.forward * 
					blackCalculator.alpha + blackCalculator.x * blackCalculator.beta);
	localvars[CR_threadIndex].result=result;
	return result;
}


//global function to retrieve the output value for an option
__global__ void getOutValOption(optionInputStruct* options, float* outputVals, int numVals, vals* localvars )
{
	int CR_blockIndex = blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x;
	int CR_threadBlockIndex = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	int CR_threadIndex = CR_blockIndex * blockDim.x * blockDim.y * blockDim.z + CR_threadBlockIndex;
	int optionNum = blockIdx.x * blockDim.x + threadIdx.x;
	localvars[CR_threadIndex].optionNum=optionNum;

	//check if within current options
	if (optionNum < numVals)
	{
		optionInputStruct threadOption = options[optionNum];
	localvars[CR_threadIndex].threadOption=threadOption;

		payoffStruct currPayoff;
		currPayoff.type = threadOption.type;
	localvars[CR_threadIndex].currPayoff.type=currPayoff.type;
		currPayoff.strike = threadOption.strike;
	localvars[CR_threadIndex].currPayoff.strike=currPayoff.strike;

		yieldTermStruct qTS;
		qTS.timeYearFraction = threadOption.t;
	localvars[CR_threadIndex].qTS.timeYearFraction=qTS.timeYearFraction;
		qTS.forward = threadOption.q;
	localvars[CR_threadIndex].qTS.forward=qTS.forward;

		yieldTermStruct rTS;
		rTS.timeYearFraction = threadOption.t;
	localvars[CR_threadIndex].rTS.timeYearFraction=rTS.timeYearFraction;
		rTS.forward = threadOption.r;
	localvars[CR_threadIndex].rTS.forward=rTS.forward;

		blackVolStruct volTS;
		volTS.timeYearFraction = threadOption.t;
	localvars[CR_threadIndex].volTS.timeYearFraction=volTS.timeYearFraction;
		volTS.volatility = threadOption.vol;
	localvars[CR_threadIndex].volTS.volatility=volTS.volatility;

		blackScholesMertStruct stochProcess;
		stochProcess.x0 = threadOption.spot;
	localvars[CR_threadIndex].stochProcess.x0=stochProcess.x0;
		stochProcess.dividendTS = qTS;
	localvars[CR_threadIndex].stochProcess.dividendTS=stochProcess.dividendTS;
		stochProcess.riskFreeTS = rTS;
	localvars[CR_threadIndex].stochProcess.riskFreeTS=stochProcess.riskFreeTS;
		stochProcess.blackVolTS = volTS;
	localvars[CR_threadIndex].stochProcess.blackVolTS=stochProcess.blackVolTS;

		optionStruct currOption;
		currOption.payoff = currPayoff;
	localvars[CR_threadIndex].currOption.payoff=currOption.payoff;
		currOption.yearFractionTime = threadOption.t;
	localvars[CR_threadIndex].currOption.yearFractionTime=currOption.yearFractionTime;
		currOption.pricingEngine = stochProcess; 

	localvars[CR_threadIndex].currOption.pricingEngine=currOption.pricingEngine;
		float variance = getBlackVolBlackVar(currOption.pricingEngine.blackVolTS,localvars);
	localvars[CR_threadIndex].variance=variance;
		float dividendDiscount = getDiscountOnDividendYield(currOption.yearFractionTime, currOption.pricingEngine.dividendTS,localvars);
	localvars[CR_threadIndex].dividendDiscount=dividendDiscount;
		float riskFreeDiscount = getDiscountOnRiskFreeRate(currOption.yearFractionTime, currOption.pricingEngine.riskFreeTS,localvars);
	localvars[CR_threadIndex].riskFreeDiscount=riskFreeDiscount;
		float spot = currOption.pricingEngine.x0; 

	localvars[CR_threadIndex].spot=spot;
		float forwardPrice = spot * dividendDiscount / riskFreeDiscount;
	localvars[CR_threadIndex].forwardPrice=forwardPrice;

		//declare the blackCalcStruct
		blackCalcStruct blackCalc;

		//initialize the calculator
		initBlackCalculator(blackCalc, currOption.payoff, forwardPrice, sqrt(variance), riskFreeDiscount,localvars);
	localvars[CR_threadIndex].blackCalc=blackCalc;

		//retrieve the results values
		float resultVal = getResultVal(blackCalc,localvars);
	localvars[CR_threadIndex].resultVal=resultVal;

		//write the resulting value to global memory
		outputVals[optionNum] = resultVal;
	}
}

#endif //BLACK_SCHOLES_ANALYTIC_ENGINE_KERNELS_CUH

