#include "hip/hip_runtime.h"
//#include "test.cu"
//blackScholesAnalyticEngineKernels.cu
//Scott Grauer-Gray
//Kernels for running black scholes using the analytic engine

#ifndef BLACK_SCHOLES_ANALYTIC_ENGINE_KERNELS_CUH
#define BLACK_SCHOLES_ANALYTIC_ENGINE_KERNELS_CUH

//declarations for the kernels
#include "blackScholesAnalyticEngineKernels.cuh"

//needed for the constants in the error function
#include "errorFunctConsts.cuh"


//device kernel to retrieve the compound factor in interestRate
__device__ float interestRateCompoundFactor(float t, yieldTermStruct currYieldTermStruct, vals* vals )
{
	return (exp((currYieldTermStruct.forward)*t));
}


//device kernel to retrieve the discount factor in interestRate
__device__ float interestRateDiscountFactor(float t, yieldTermStruct currYieldTermStruct, vals* vals )
{
	return 1.0f / interestRateCompoundFactor(t, currYieldTermStruct,vals);
}


//device function to get the variance of the black volatility function
__device__ float getBlackVolBlackVar(blackVolStruct volTS, vals* vals )
{
	float vol = volTS.volatility;
	vals[CR_threadIndex].vol=vol;
	return vol*vol*volTS.timeYearFraction;
}


//device function to get the discount on a dividend yield
__device__ float getDiscountOnDividendYield(float yearFraction, yieldTermStruct dividendYieldTermStruct, vals* vals )
{
	float intDiscountFactor = interestRateDiscountFactor(yearFraction, dividendYieldTermStruct,vals);
	vals[CR_threadIndex].intDiscountFactor=intDiscountFactor;
	return intDiscountFactor;
}


//device function to get the discount on the risk free rate
__device__ float getDiscountOnRiskFreeRate(float yearFraction, yieldTermStruct riskFreeRateYieldTermStruct, vals* vals )
{
	return interestRateDiscountFactor(yearFraction, riskFreeRateYieldTermStruct,vals);
}


//device kernel to run the error function
__device__ float errorFunct(normalDistStruct normDist, float x, vals* vals )
{
	float R,S,P,Q,s,y,z,r, ax;

    ax = fabs(x);
	vals[CR_threadIndex].ax=ax;

    if(ax < 0.84375) 
	{      
        if(ax < 3.7252902984e-09) 
		{ 
                if (ax < DBL_MIN*16)
                    return 0.125*(8.0*x+ (ERROR_FUNCT_efx8)*x);  /*avoid underflow */
	            return x + (ERROR_FUNCT_efx)*x;
        }
		z = x*x;
	vals[CR_threadIndex].z=z;
        r = ERROR_FUNCT_pp0+z*(ERROR_FUNCT_pp1+z*(ERROR_FUNCT_pp2+z*(ERROR_FUNCT_pp3+z*ERROR_FUNCT_pp4)));
	vals[CR_threadIndex].r=r;
        s = ERROR_FUNCT_one+z*(ERROR_FUNCT_qq1+z*(ERROR_FUNCT_qq2+z*(ERROR_FUNCT_qq3+z*(ERROR_FUNCT_qq4+z*ERROR_FUNCT_qq5))));
	vals[CR_threadIndex].s=s;
        y = r/s;
	vals[CR_threadIndex].y=y;
        return x + x*y;
    }
    if(ax <1.25) 
	{      
        s = ax-ERROR_FUNCT_one;
	vals[CR_threadIndex].s=s;
        P = ERROR_FUNCT_pa0+s*(ERROR_FUNCT_pa1+s*(ERROR_FUNCT_pa2+s*(ERROR_FUNCT_pa3+s*(ERROR_FUNCT_pa4+s*(ERROR_FUNCT_pa5+s*ERROR_FUNCT_pa6)))));
	vals[CR_threadIndex].P=P;
        Q = ERROR_FUNCT_one+s*(ERROR_FUNCT_qa1+s*(ERROR_FUNCT_qa2+s*(ERROR_FUNCT_qa3+s*(ERROR_FUNCT_qa4+s*(ERROR_FUNCT_qa5+s*ERROR_FUNCT_qa6)))));
	vals[CR_threadIndex].Q=Q;
        if(x>=0) return ERROR_FUNCT_erx + P/Q; else return -1*ERROR_FUNCT_erx - P/Q;
    }
    if (ax >= 6) 
	{      
        if(x>=0) 
			return ERROR_FUNCT_one-ERROR_FUNCT_tiny; 
		else 
			return ERROR_FUNCT_tiny-ERROR_FUNCT_one;
    }

    /* Starts to lose accuracy when ax~5 */
    s = ERROR_FUNCT_one/(ax*ax);
	vals[CR_threadIndex].s=s;

    if(ax < 2.85714285714285) { /* |x| < 1/0.35 */
        R = ERROR_FUNCT_ra0+s*(ERROR_FUNCT_ra1+s*(ERROR_FUNCT_ra2+s*(ERROR_FUNCT_ra3+s*(ERROR_FUNCT_ra4+s*(ERROR_FUNCT_ra5+s*(ERROR_FUNCT_ra6+s*ERROR_FUNCT_ra7))))));
	vals[CR_threadIndex].R=R;
        S = ERROR_FUNCT_one+s*(ERROR_FUNCT_sa1+s*(ERROR_FUNCT_sa2+s*(ERROR_FUNCT_sa3+s*(ERROR_FUNCT_sa4+s*(ERROR_FUNCT_sa5+s*(ERROR_FUNCT_sa6+s*(ERROR_FUNCT_sa7+s*ERROR_FUNCT_sa8)))))));
	vals[CR_threadIndex].S=S;
    } else {    /* |x| >= 1/0.35 */
        R=ERROR_FUNCT_rb0+s*(ERROR_FUNCT_rb1+s*(ERROR_FUNCT_rb2+s*(ERROR_FUNCT_rb3+s*(ERROR_FUNCT_rb4+s*(ERROR_FUNCT_rb5+s*ERROR_FUNCT_rb6)))));
	vals[CR_threadIndex].R=R;
        S=ERROR_FUNCT_one+s*(ERROR_FUNCT_sb1+s*(ERROR_FUNCT_sb2+s*(ERROR_FUNCT_sb3+s*(ERROR_FUNCT_sb4+s*(ERROR_FUNCT_sb5+s*(ERROR_FUNCT_sb6+s*ERROR_FUNCT_sb7))))));
	vals[CR_threadIndex].S=S;
    }

    r = exp( -ax*ax-0.5625 +R/S);
	vals[CR_threadIndex].r=r;
    if(x>=0) 
		return ERROR_FUNCT_one-r/ax; 
	else 
		return r/ax-ERROR_FUNCT_one;
}



//device kernel to run the operator function in cumulative normal distribution
__device__ float cumNormDistOp(normalDistStruct normDist, float z, vals* vals )
{
	z = (z - normDist.average) / normDist.sigma;
	vals[CR_threadIndex].z=z;
    	float result = 0.5 * ( 1.0 + errorFunct(normDist, z*M_SQRT_2 ,vals) );
	vals[CR_threadIndex].result=result;
	return result;
}


//device kernel to run the gaussian function in the normal distribution
__device__ float gaussianFunctNormDist(normalDistStruct normDist, float x, vals* vals )
{
	float deltax = x - normDist.average;
	vals[CR_threadIndex].deltax=deltax;
	float exponent = -(deltax*deltax)/normDist.denominator;
	vals[CR_threadIndex].exponent=exponent;

    	// debian alpha had some strange problem in the very-low range
    	return exponent <= -690.0 ? 0.0 :  // exp(x) < 1.0e-300 anyway
            normDist.normalizationFactor * exp(exponent);
}


//device kernel to retrieve the derivative in a cumulative normal distribution
__device__ float cumNormDistDeriv(normalDistStruct normDist, float x, vals* vals )
{
	float xn = (x - normDist.average) / normDist.sigma;
	vals[CR_threadIndex].xn=xn;
    	return gaussianFunctNormDist(normDist, xn,vals) / normDist.sigma;
}


//device function to initialize the cumulative normal distribution structure
__device__ void initCumNormDist(normalDistStruct& currCumNormDist, vals* vals )
{
	currCumNormDist.average = 0.0f;
	vals[CR_threadIndex].currCumNormDist->average=currCumNormDist->average;
	currCumNormDist.sigma = 1.0f;
	vals[CR_threadIndex].currCumNormDist->sigma=currCumNormDist->sigma;
	currCumNormDist.normalizationFactor = M_SQRT_2*M_1_SQRTPI/currCumNormDist.sigma;
	vals[CR_threadIndex].currCumNormDist->normalizationFactor=currCumNormDist->normalizationFactor;
    	currCumNormDist.derNormalizationFactor = currCumNormDist.sigma*currCumNormDist.sigma;
	vals[CR_threadIndex].currCumNormDist->derNormalizationFactor=currCumNormDist->derNormalizationFactor;
    	currCumNormDist.denominator = 2.0*currCumNormDist.derNormalizationFactor;
	vals[CR_threadIndex].currCumNormDist->denominator=currCumNormDist->denominator;
}


//device function to initialize variable in the black calculator
__device__ void initBlackCalcVars(blackCalcStruct& blackCalculator, payoffStruct payoff, vals* vals )
{
	blackCalculator.d1 = log(blackCalculator.forward / blackCalculator.strike)/blackCalculator.stdDev + 0.5*blackCalculator.stdDev;
	blackCalculator.d2 = blackCalculator.d1 - blackCalculator.stdDev;

	//initialize the cumulative normal distribution structure
	normalDistStruct currCumNormDist;
	initCumNormDist(currCumNormDist,vals);
	vals[CR_threadIndex].currCumNormDist=currCumNormDist;
                
	blackCalculator.cum_d1 = cumNormDistOp(currCumNormDist, blackCalculator.d1,vals);
	blackCalculator.cum_d2 = cumNormDistOp(currCumNormDist, blackCalculator.d2,vals);
	blackCalculator.n_d1 = cumNormDistDeriv(currCumNormDist, blackCalculator.d1,vals);
	blackCalculator.n_d2 = cumNormDistDeriv(currCumNormDist, blackCalculator.d2,vals);

	blackCalculator.x = payoff.strike;
	blackCalculator.DxDstrike = 1.0;

	// the following one will probably disappear as soon as
	// super-share will be properly handled
	blackCalculator.DxDs = 0.0;

	// this part is always executed.
	// in case of plain-vanilla payoffs, it is also the only part
	// which is executed.
	switch (payoff.type) 
	{
		case CALL:
		    blackCalculator.alpha     =  blackCalculator.cum_d1;//  N(d1)
		    blackCalculator.DalphaDd1 =    blackCalculator.n_d1;//  n(d1)
		    blackCalculator.beta      = -1.0f*blackCalculator.cum_d2;// -N(d2)
		    blackCalculator.DbetaDd2  = -1.0f*blackCalculator.n_d2;// -n(d2)
		    break;
		case PUT:
		    blackCalculator.alpha     = -1.0+blackCalculator.cum_d1;// -N(-d1)
		    blackCalculator.DalphaDd1 =        blackCalculator.n_d1;//  n( d1)
		    blackCalculator.beta      =  1.0-blackCalculator.cum_d2;//  N(-d2)
		    blackCalculator.DbetaDd2  =     -1.0f* blackCalculator.n_d2;// -n( d2)
		    break;
	}
}


//device function to initialize the black calculator
__device__ void initBlackCalculator(blackCalcStruct& blackCalc, payoffStruct payoff, float forwardPrice, float stdDev, float riskFreeDiscount, vals* vals )
{
	blackCalc.strike = payoff.strike;
	vals[CR_threadIndex].blackCalc->strike=blackCalc->strike;
	blackCalc.forward = forwardPrice;
	vals[CR_threadIndex].blackCalc->forward=blackCalc->forward;
	blackCalc.stdDev = stdDev;
	vals[CR_threadIndex].blackCalc->stdDev=blackCalc->stdDev;
	blackCalc.discount = riskFreeDiscount;
	vals[CR_threadIndex].blackCalc->discount=blackCalc->discount;
	blackCalc.variance = stdDev * stdDev;
	vals[CR_threadIndex].blackCalc->variance=blackCalc->variance;

	initBlackCalcVars(blackCalc, payoff,vals);
	vals[CR_threadIndex].blackCalc=blackCalc;
}


//device function to retrieve the output resulting value
__device__ float getResultVal(blackCalcStruct blackCalculator, vals* vals )
{
	float result = blackCalculator.discount * (blackCalculator.forward * 
					blackCalculator.alpha + blackCalculator.x * blackCalculator.beta);
	vals[CR_threadIndex].result=result;
	return result;
}


//global function to retrieve the output value for an option
__global__ void getOutValOption(optionInputStruct* options, float* outputVals, int numVals, vals* vals )
{
	int CR_blockIndex = blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x;
	int CR_threadBlockIndex = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	int CR_threadIndex = CR_blockIndex * blockDim.x * blockDim.y * blockDim.z + CR_threadBlockIndex;
	int optionNum = blockIdx.x * blockDim.x + threadIdx.x;
	vals[CR_threadIndex].optionNum=optionNum;

	//check if within current options
	if (optionNum < numVals)
	{
		optionInputStruct threadOption = options[optionNum];
	vals[CR_threadIndex].threadOption=threadOption;

		payoffStruct currPayoff;
		currPayoff.type = threadOption.type;
	vals[CR_threadIndex].currPayoff.type=currPayoff.type;
		currPayoff.strike = threadOption.strike;
	vals[CR_threadIndex].currPayoff.strike=currPayoff.strike;

		yieldTermStruct qTS;
		qTS.timeYearFraction = threadOption.t;
	vals[CR_threadIndex].qTS.timeYearFraction=qTS.timeYearFraction;
		qTS.forward = threadOption.q;
	vals[CR_threadIndex].qTS.forward=qTS.forward;

		yieldTermStruct rTS;
		rTS.timeYearFraction = threadOption.t;
	vals[CR_threadIndex].rTS.timeYearFraction=rTS.timeYearFraction;
		rTS.forward = threadOption.r;
	vals[CR_threadIndex].rTS.forward=rTS.forward;

		blackVolStruct volTS;
		volTS.timeYearFraction = threadOption.t;
	vals[CR_threadIndex].volTS.timeYearFraction=volTS.timeYearFraction;
		volTS.volatility = threadOption.vol;
	vals[CR_threadIndex].volTS.volatility=volTS.volatility;

		blackScholesMertStruct stochProcess;
		stochProcess.x0 = threadOption.spot;
	vals[CR_threadIndex].stochProcess.x0=stochProcess.x0;
		stochProcess.dividendTS = qTS;
	vals[CR_threadIndex].stochProcess.dividendTS=stochProcess.dividendTS;
		stochProcess.riskFreeTS = rTS;
	vals[CR_threadIndex].stochProcess.riskFreeTS=stochProcess.riskFreeTS;
		stochProcess.blackVolTS = volTS;
	vals[CR_threadIndex].stochProcess.blackVolTS=stochProcess.blackVolTS;

		optionStruct currOption;
		currOption.payoff = currPayoff;
	vals[CR_threadIndex].currOption.payoff=currOption.payoff;
		currOption.yearFractionTime = threadOption.t;
	vals[CR_threadIndex].currOption.yearFractionTime=currOption.yearFractionTime;
		currOption.pricingEngine = stochProcess; 

	vals[CR_threadIndex].currOption.pricingEngine=currOption.pricingEngine;
		float variance = getBlackVolBlackVar(currOption.pricingEngine.blackVolTS,vals);
	vals[CR_threadIndex].variance=variance;
		float dividendDiscount = getDiscountOnDividendYield(currOption.yearFractionTime, currOption.pricingEngine.dividendTS,vals);
	vals[CR_threadIndex].dividendDiscount=dividendDiscount;
		float riskFreeDiscount = getDiscountOnRiskFreeRate(currOption.yearFractionTime, currOption.pricingEngine.riskFreeTS,vals);
	vals[CR_threadIndex].riskFreeDiscount=riskFreeDiscount;
		float spot = currOption.pricingEngine.x0; 

	vals[CR_threadIndex].spot=spot;
		float forwardPrice = spot * dividendDiscount / riskFreeDiscount;
	vals[CR_threadIndex].forwardPrice=forwardPrice;

		//declare the blackCalcStruct
		blackCalcStruct blackCalc;

		//initialize the calculator
		initBlackCalculator(blackCalc, currOption.payoff, forwardPrice, sqrt(variance), riskFreeDiscount,vals);
	vals[CR_threadIndex].blackCalc=blackCalc;

		//retrieve the results values
		float resultVal = getResultVal(blackCalc,vals);
	vals[CR_threadIndex].resultVal=resultVal;

		//write the resulting value to global memory
		outputVals[optionNum] = resultVal;
	}
}

#endif //BLACK_SCHOLES_ANALYTIC_ENGINE_KERNELS_CUH

