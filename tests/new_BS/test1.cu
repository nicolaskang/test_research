#include "hip/hip_runtime.h"
#include "test.cu"
//blackScholesAnalyticEngineKernels.cu
//Scott Grauer-Gray
//Kernels for running black scholes using the analytic engine

#ifndef BLACK_SCHOLES_ANALYTIC_ENGINE_KERNELS_CUH
#define BLACK_SCHOLES_ANALYTIC_ENGINE_KERNELS_CUH

//declarations for the kernels
#include "blackScholesAnalyticEngineKernels.cuh"

//needed for the constants in the error function
#include "errorFunctConsts.cuh"


//device kernel to retrieve the compound factor in interestRate
__device__ float interestRateCompoundFactor(float t, yieldTermStruct currYieldTermStruct, vals* vals )
{
	return (exp((currYieldTermStruct.forward)*t));
}


//device kernel to retrieve the discount factor in interestRate
__device__ float interestRateDiscountFactor(float t, yieldTermStruct currYieldTermStruct, vals* vals )
{
	return 1.0f / interestRateCompoundFactor(t, currYieldTermStruct,vals);
}


//device function to get the variance of the black volatility function
__device__ float getBlackVolBlackVar(blackVolStruct volTS, vals* vals )
{
	float vol = volTS.volatility;
	vals!@#$^&*(.vol=vol;
	return vol*vol*volTS.timeYearFraction;
}


//device function to get the discount on a dividend yield
__device__ float getDiscountOnDividendYield(float yearFraction, yieldTermStruct dividendYieldTermStruct, vals* vals )
{
	float intDiscountFactor = interestRateDiscountFactor(yearFraction, dividendYieldTermStruct,vals);
	vals!@#$^&*(.intDiscountFactor=intDiscountFactor;
	return intDiscountFactor;
}


//device function to get the discount on the risk free rate
__device__ float getDiscountOnRiskFreeRate(float yearFraction, yieldTermStruct riskFreeRateYieldTermStruct, vals* vals )
{
	return interestRateDiscountFactor(yearFraction, riskFreeRateYieldTermStruct,vals);
}


//device kernel to run the error function
__device__ float errorFunct(normalDistStruct normDist, float x, vals* vals )
{
	float R,S,P,Q,s,y,z,r, ax;

    ax = fabs(x);
	vals!@#$^&*(.ax=ax;

    if(ax < 0.84375) 
	{      
        if(ax < 3.7252902984e-09) 
		{ 
                if (ax < DBL_MIN*16)
                    return 0.125*(8.0*x+ (ERROR_FUNCT_efx8)*x);  /*avoid underflow */
	            return x + (ERROR_FUNCT_efx)*x;
        }
		z = x*x;
	vals!@#$^&*(.z=z;
        r = ERROR_FUNCT_pp0+z*(ERROR_FUNCT_pp1+z*(ERROR_FUNCT_pp2+z*(ERROR_FUNCT_pp3+z*ERROR_FUNCT_pp4)));
	vals!@#$^&*(.r=r;
        s = ERROR_FUNCT_one+z*(ERROR_FUNCT_qq1+z*(ERROR_FUNCT_qq2+z*(ERROR_FUNCT_qq3+z*(ERROR_FUNCT_qq4+z*ERROR_FUNCT_qq5))));
	vals!@#$^&*(.s=s;
        y = r/s;
	vals!@#$^&*(.y=y;
        return x + x*y;
    }
    if(ax <1.25) 
	{      
        s = ax-ERROR_FUNCT_one;
	vals!@#$^&*(.s=s;
        P = ERROR_FUNCT_pa0+s*(ERROR_FUNCT_pa1+s*(ERROR_FUNCT_pa2+s*(ERROR_FUNCT_pa3+s*(ERROR_FUNCT_pa4+s*(ERROR_FUNCT_pa5+s*ERROR_FUNCT_pa6)))));
	vals!@#$^&*(.P=P;
        Q = ERROR_FUNCT_one+s*(ERROR_FUNCT_qa1+s*(ERROR_FUNCT_qa2+s*(ERROR_FUNCT_qa3+s*(ERROR_FUNCT_qa4+s*(ERROR_FUNCT_qa5+s*ERROR_FUNCT_qa6)))));
	vals!@#$^&*(.Q=Q;
        if(x>=0) return ERROR_FUNCT_erx + P/Q; else return -1*ERROR_FUNCT_erx - P/Q;
    }
    if (ax >= 6) 
	{      
        if(x>=0) 
			return ERROR_FUNCT_one-ERROR_FUNCT_tiny; 
		else 
			return ERROR_FUNCT_tiny-ERROR_FUNCT_one;
    }

    /* Starts to lose accuracy when ax~5 */
    s = ERROR_FUNCT_one/(ax*ax);
	vals!@#$^&*(.s=s;

    if(ax < 2.85714285714285) { /* |x| < 1/0.35 */
        R = ERROR_FUNCT_ra0+s*(ERROR_FUNCT_ra1+s*(ERROR_FUNCT_ra2+s*(ERROR_FUNCT_ra3+s*(ERROR_FUNCT_ra4+s*(ERROR_FUNCT_ra5+s*(ERROR_FUNCT_ra6+s*ERROR_FUNCT_ra7))))));
	vals!@#$^&*(.R=R;
        S = ERROR_FUNCT_one+s*(ERROR_FUNCT_sa1+s*(ERROR_FUNCT_sa2+s*(ERROR_FUNCT_sa3+s*(ERROR_FUNCT_sa4+s*(ERROR_FUNCT_sa5+s*(ERROR_FUNCT_sa6+s*(ERROR_FUNCT_sa7+s*ERROR_FUNCT_sa8)))))));
	vals!@#$^&*(.S=S;
    } else {    /* |x| >= 1/0.35 */
        R=ERROR_FUNCT_rb0+s*(ERROR_FUNCT_rb1+s*(ERROR_FUNCT_rb2+s*(ERROR_FUNCT_rb3+s*(ERROR_FUNCT_rb4+s*(ERROR_FUNCT_rb5+s*ERROR_FUNCT_rb6)))));
	vals!@#$^&*(.R=R;
        S=ERROR_FUNCT_one+s*(ERROR_FUNCT_sb1+s*(ERROR_FUNCT_sb2+s*(ERROR_FUNCT_sb3+s*(ERROR_FUNCT_sb4+s*(ERROR_FUNCT_sb5+s*(ERROR_FUNCT_sb6+s*ERROR_FUNCT_sb7))))));
	vals!@#$^&*(.S=S;
    }

    r = exp( -ax*ax-0.5625 +R/S);
	vals!@#$^&*(.r=r;
    if(x>=0) 
		return ERROR_FUNCT_one-r/ax; 
	else 
		return r/ax-ERROR_FUNCT_one;
}



//device kernel to run the operator function in cumulative normal distribution
__device__ float cumNormDistOp(normalDistStruct normDist, float z, vals* vals )
{
	z = (z - normDist.average) / normDist.sigma;
	vals!@#$^&*(.z=z;
    	float result = 0.5 * ( 1.0 + errorFunct(normDist, z*M_SQRT_2 ,vals) );
	vals!@#$^&*(.result=result;
	return result;
}


//device kernel to run the gaussian function in the normal distribution
__device__ float gaussianFunctNormDist(normalDistStruct normDist, float x, vals* vals )
{
	float deltax = x - normDist.average;
	vals!@#$^&*(.deltax=deltax;
	float exponent = -(deltax*deltax)/normDist.denominator;
	vals!@#$^&*(.exponent=exponent;

    	// debian alpha had some strange problem in the very-low range
    	return exponent <= -690.0 ? 0.0 :  // exp(x) < 1.0e-300 anyway
            normDist.normalizationFactor * exp(exponent);
}


//device kernel to retrieve the derivative in a cumulative normal distribution
__device__ float cumNormDistDeriv(normalDistStruct normDist, float x, vals* vals )
{
	float xn = (x - normDist.average) / normDist.sigma;
	vals!@#$^&*(.xn=xn;
    	return gaussianFunctNormDist(normDist, xn,vals) / normDist.sigma;
}


//device function to initialize the cumulative normal distribution structure
__device__ void initCumNormDist(normalDistStruct& currCumNormDist, vals* vals )
{
	currCumNormDist.average = 0.0f;
	vals!@#$^&*(.currCumNormDist->average=currCumNormDist->average;
	currCumNormDist.sigma = 1.0f;
	vals!@#$^&*(.currCumNormDist->sigma=currCumNormDist->sigma;
	currCumNormDist.normalizationFactor = M_SQRT_2*M_1_SQRTPI/currCumNormDist.sigma;
	vals!@#$^&*(.currCumNormDist->normalizationFactor=currCumNormDist->normalizationFactor;
    	currCumNormDist.derNormalizationFactor = currCumNormDist.sigma*currCumNormDist.sigma;
	vals!@#$^&*(.currCumNormDist->derNormalizationFactor=currCumNormDist->derNormalizationFactor;
    	currCumNormDist.denominator = 2.0*currCumNormDist.derNormalizationFactor;
	vals!@#$^&*(.currCumNormDist->denominator=currCumNormDist->denominator;
}


//device function to initialize variable in the black calculator
__device__ void initBlackCalcVars(blackCalcStruct& blackCalculator, payoffStruct payoff, vals* vals )
{
	blackCalculator.d1 = log(blackCalculator.forward / blackCalculator.strike)/blackCalculator.stdDev + 0.5*blackCalculator.stdDev;
	blackCalculator.d2 = blackCalculator.d1 - blackCalculator.stdDev;

	//initialize the cumulative normal distribution structure
	normalDistStruct currCumNormDist;
	initCumNormDist(currCumNormDist,vals);
	vals!@#$^&*(.currCumNormDist=currCumNormDist;
                
	blackCalculator.cum_d1 = cumNormDistOp(currCumNormDist, blackCalculator.d1,vals);
	blackCalculator.cum_d2 = cumNormDistOp(currCumNormDist, blackCalculator.d2,vals);
	blackCalculator.n_d1 = cumNormDistDeriv(currCumNormDist, blackCalculator.d1,vals);
	blackCalculator.n_d2 = cumNormDistDeriv(currCumNormDist, blackCalculator.d2,vals);

	blackCalculator.x = payoff.strike;
	blackCalculator.DxDstrike = 1.0;

	// the following one will probably disappear as soon as
	// super-share will be properly handled
	blackCalculator.DxDs = 0.0;

	// this part is always executed.
	// in case of plain-vanilla payoffs, it is also the only part
	// which is executed.
	switch (payoff.type) 
	{
		case CALL:
		    blackCalculator.alpha     =  blackCalculator.cum_d1;//  N(d1)
		    blackCalculator.DalphaDd1 =    blackCalculator.n_d1;//  n(d1)
		    blackCalculator.beta      = -1.0f*blackCalculator.cum_d2;// -N(d2)
		    blackCalculator.DbetaDd2  = -1.0f*blackCalculator.n_d2;// -n(d2)
		    break;
		case PUT:
		    blackCalculator.alpha     = -1.0+blackCalculator.cum_d1;// -N(-d1)
		    blackCalculator.DalphaDd1 =        blackCalculator.n_d1;//  n( d1)
		    blackCalculator.beta      =  1.0-blackCalculator.cum_d2;//  N(-d2)
		    blackCalculator.DbetaDd2  =     -1.0f* blackCalculator.n_d2;// -n( d2)
		    break;
	}
}


//device function to initialize the black calculator
__device__ void initBlackCalculator(blackCalcStruct& blackCalc, payoffStruct payoff, float forwardPrice, float stdDev, float riskFreeDiscount, vals* vals )
{
	blackCalc.strike = payoff.strike;
	vals!@#$^&*(.blackCalc->strike=blackCalc->strike;
	blackCalc.forward = forwardPrice;
	vals!@#$^&*(.blackCalc->forward=blackCalc->forward;
	blackCalc.stdDev = stdDev;
	vals!@#$^&*(.blackCalc->stdDev=blackCalc->stdDev;
	blackCalc.discount = riskFreeDiscount;
	vals!@#$^&*(.blackCalc->discount=blackCalc->discount;
	blackCalc.variance = stdDev * stdDev;
	vals!@#$^&*(.blackCalc->variance=blackCalc->variance;

	initBlackCalcVars(blackCalc, payoff,vals);
	vals!@#$^&*(.blackCalc=blackCalc;
}


//device function to retrieve the output resulting value
__device__ float getResultVal(blackCalcStruct blackCalculator, vals* vals )
{
	float result = blackCalculator.discount * (blackCalculator.forward * 
					blackCalculator.alpha + blackCalculator.x * blackCalculator.beta);
	vals!@#$^&*(.result=result;
	return result;
}


//global function to retrieve the output value for an option
__global__ void getOutValOption(optionInputStruct* options, float* outputVals, int numVals, vals* vals )
{
	int optionNum = blockIdx.x * blockDim.x + threadIdx.x;
	vals!@#$^&*(.optionNum=optionNum;

	//check if within current options
	if (optionNum < numVals)
	{
		optionInputStruct threadOption = options[optionNum];
	vals!@#$^&*(.threadOption=threadOption;

		payoffStruct currPayoff;
		currPayoff.type = threadOption.type;
	vals!@#$^&*(.currPayoff.type=currPayoff.type;
		currPayoff.strike = threadOption.strike;
	vals!@#$^&*(.currPayoff.strike=currPayoff.strike;

		yieldTermStruct qTS;
		qTS.timeYearFraction = threadOption.t;
	vals!@#$^&*(.qTS.timeYearFraction=qTS.timeYearFraction;
		qTS.forward = threadOption.q;
	vals!@#$^&*(.qTS.forward=qTS.forward;

		yieldTermStruct rTS;
		rTS.timeYearFraction = threadOption.t;
	vals!@#$^&*(.rTS.timeYearFraction=rTS.timeYearFraction;
		rTS.forward = threadOption.r;
	vals!@#$^&*(.rTS.forward=rTS.forward;

		blackVolStruct volTS;
		volTS.timeYearFraction = threadOption.t;
	vals!@#$^&*(.volTS.timeYearFraction=volTS.timeYearFraction;
		volTS.volatility = threadOption.vol;
	vals!@#$^&*(.volTS.volatility=volTS.volatility;

		blackScholesMertStruct stochProcess;
		stochProcess.x0 = threadOption.spot;
	vals!@#$^&*(.stochProcess.x0=stochProcess.x0;
		stochProcess.dividendTS = qTS;
	vals!@#$^&*(.stochProcess.dividendTS=stochProcess.dividendTS;
		stochProcess.riskFreeTS = rTS;
	vals!@#$^&*(.stochProcess.riskFreeTS=stochProcess.riskFreeTS;
		stochProcess.blackVolTS = volTS;
	vals!@#$^&*(.stochProcess.blackVolTS=stochProcess.blackVolTS;

		optionStruct currOption;
		currOption.payoff = currPayoff;
	vals!@#$^&*(.currOption.payoff=currOption.payoff;
		currOption.yearFractionTime = threadOption.t;
	vals!@#$^&*(.currOption.yearFractionTime=currOption.yearFractionTime;
		currOption.pricingEngine = stochProcess; 

	vals!@#$^&*(.currOption.pricingEngine=currOption.pricingEngine;
		float variance = getBlackVolBlackVar(currOption.pricingEngine.blackVolTS,vals);
	vals!@#$^&*(.variance=variance;
		float dividendDiscount = getDiscountOnDividendYield(currOption.yearFractionTime, currOption.pricingEngine.dividendTS,vals);
	vals!@#$^&*(.dividendDiscount=dividendDiscount;
		float riskFreeDiscount = getDiscountOnRiskFreeRate(currOption.yearFractionTime, currOption.pricingEngine.riskFreeTS,vals);
	vals!@#$^&*(.riskFreeDiscount=riskFreeDiscount;
		float spot = currOption.pricingEngine.x0; 

	vals!@#$^&*(.spot=spot;
		float forwardPrice = spot * dividendDiscount / riskFreeDiscount;
	vals!@#$^&*(.forwardPrice=forwardPrice;

		//declare the blackCalcStruct
		blackCalcStruct blackCalc;

		//initialize the calculator
		initBlackCalculator(blackCalc, currOption.payoff, forwardPrice, sqrt(variance), riskFreeDiscount,vals);
	vals!@#$^&*(.blackCalc=blackCalc;

		//retrieve the results values
		float resultVal = getResultVal(blackCalc,vals);
	vals!@#$^&*(.resultVal=resultVal;

		//write the resulting value to global memory
		outputVals[optionNum] = resultVal;
	}
}

#endif //BLACK_SCHOLES_ANALYTIC_ENGINE_KERNELS_CUH

