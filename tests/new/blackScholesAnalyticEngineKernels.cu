#include "hip/hip_runtime.h"
 #ifndef BLACK_SCHOLES_ANALYTIC_ENGINE_KERNELS_CUH
#define BLACK_SCHOLES_ANALYTIC_ENGINE_KERNELS_CUH
 #include "blackScholesAnalyticEngineKernels.cuh"
 #include "errorFunctConsts.cuh"
 __device__ float interestRateCompoundFactor(float t, yieldTermStruct currYieldTermStruct)
{
return (exp((currYieldTermStruct.forward)*t));
}
 __device__ float interestRateDiscountFactor(float t, yieldTermStruct currYieldTermStruct)
{
return 1.0f / interestRateCompoundFactor(t, currYieldTermStruct);
}
 __device__ float getBlackVolBlackVar(blackVolStruct volTS)
{
float vol = volTS.volatility;
return vol*vol*volTS.timeYearFraction;
}
 __device__ float getDiscountOnDividendYield(float yearFraction, yieldTermStruct dividendYieldTermStruct)
{
float intDiscountFactor = interestRateDiscountFactor(yearFraction, dividendYieldTermStruct);
return intDiscountFactor;
}
 __device__ float getDiscountOnRiskFreeRate(float yearFraction, yieldTermStruct riskFreeRateYieldTermStruct)
{
return interestRateDiscountFactor(yearFraction, riskFreeRateYieldTermStruct);
}
 __device__ float errorFunct(normalDistStruct normDist, float x)
{
float R,S,P,Q,s,y,z,r, ax;
 ax = fabs(x);
 if(ax < 0.84375) 
{ 
 if(ax < 3.7252902984e-09) 
{ 
 if (ax < DBL_MIN*16)
 return 0.125*(8.0*x+ (ERROR_FUNCT_efx8)*x); /*avoid underflow */
 return x + (ERROR_FUNCT_efx)*x;
 }
z = x*x;
 r = ERROR_FUNCT_pp0+z*(ERROR_FUNCT_pp1+z*(ERROR_FUNCT_pp2+z*(ERROR_FUNCT_pp3+z*ERROR_FUNCT_pp4)));
 s = ERROR_FUNCT_one+z*(ERROR_FUNCT_qq1+z*(ERROR_FUNCT_qq2+z*(ERROR_FUNCT_qq3+z*(ERROR_FUNCT_qq4+z*ERROR_FUNCT_qq5))));
 y = r/s;
 return x + x*y;
 }
 if(ax <1.25) 
{ 
 s = ax-ERROR_FUNCT_one;
 P = ERROR_FUNCT_pa0+s*(ERROR_FUNCT_pa1+s*(ERROR_FUNCT_pa2+s*(ERROR_FUNCT_pa3+s*(ERROR_FUNCT_pa4+s*(ERROR_FUNCT_pa5+s*ERROR_FUNCT_pa6)))));
 Q = ERROR_FUNCT_one+s*(ERROR_FUNCT_qa1+s*(ERROR_FUNCT_qa2+s*(ERROR_FUNCT_qa3+s*(ERROR_FUNCT_qa4+s*(ERROR_FUNCT_qa5+s*ERROR_FUNCT_qa6)))));
 if(x>=0) return ERROR_FUNCT_erx + P/Q; else return -1*ERROR_FUNCT_erx - P/Q;
 }
 if (ax >= 6) 
{ 
 if(x>=0) 
return ERROR_FUNCT_one-ERROR_FUNCT_tiny; 
else return ERROR_FUNCT_tiny-ERROR_FUNCT_one;
 }
 /* Starts to lose accuracy when ax~5 */ s = ERROR_FUNCT_one/(ax*ax);
 if(ax < 2.85714285714285) { /* |x| < 1/0.35 */
 R = ERROR_FUNCT_ra0+s*(ERROR_FUNCT_ra1+s*(ERROR_FUNCT_ra2+s*(ERROR_FUNCT_ra3+s*(ERROR_FUNCT_ra4+s*(ERROR_FUNCT_ra5+s*(ERROR_FUNCT_ra6+s*ERROR_FUNCT_ra7))))));
 S = ERROR_FUNCT_one+s*(ERROR_FUNCT_sa1+s*(ERROR_FUNCT_sa2+s*(ERROR_FUNCT_sa3+s*(ERROR_FUNCT_sa4+s*(ERROR_FUNCT_sa5+s*(ERROR_FUNCT_sa6+s*(ERROR_FUNCT_sa7+s*ERROR_FUNCT_sa8)))))));
 } else { /* |x| >= 1/0.35 */
 R=ERROR_FUNCT_rb0+s*(ERROR_FUNCT_rb1+s*(ERROR_FUNCT_rb2+s*(ERROR_FUNCT_rb3+s*(ERROR_FUNCT_rb4+s*(ERROR_FUNCT_rb5+s*ERROR_FUNCT_rb6)))));
 S=ERROR_FUNCT_one+s*(ERROR_FUNCT_sb1+s*(ERROR_FUNCT_sb2+s*(ERROR_FUNCT_sb3+s*(ERROR_FUNCT_sb4+s*(ERROR_FUNCT_sb5+s*(ERROR_FUNCT_sb6+s*ERROR_FUNCT_sb7))))));
 }
 r = exp( -ax*ax-0.5625 +R/S);
 if(x>=0) 
return ERROR_FUNCT_one-r/ax; 
else return r/ax-ERROR_FUNCT_one;
}
 __device__ float cumNormDistOp(normalDistStruct normDist, float z)
{
z = (z - normDist.average) / normDist.sigma;
 float result = 0.5 * ( 1.0 + errorFunct(normDist, z*M_SQRT_2 ) );
return result;
}
 __device__ float gaussianFunctNormDist(normalDistStruct normDist, float x)
{
float deltax = x - normDist.average;
float exponent = -(deltax*deltax)/normDist.denominator;
 return exponent <= -690.0 ? 0.0 : normDist.normalizationFactor * exp(exponent);
}
 __device__ float cumNormDistDeriv(normalDistStruct normDist, float x)
{
float xn = (x - normDist.average) / normDist.sigma;
 return gaussianFunctNormDist(normDist, xn) / normDist.sigma;
}
 __device__ void initCumNormDist(normalDistStruct& currCumNormDist)
{
currCumNormDist.average = 0.0f;
currCumNormDist.sigma = 1.0f;
currCumNormDist.normalizationFactor = M_SQRT_2*M_1_SQRTPI/currCumNormDist.sigma;
 currCumNormDist.derNormalizationFactor = currCumNormDist.sigma*currCumNormDist.sigma;
 currCumNormDist.denominator = 2.0*currCumNormDist.derNormalizationFactor;
}
 __device__ void initBlackCalcVars(blackCalcStruct& blackCalculator, payoffStruct payoff)
{
blackCalculator.d1 = log(blackCalculator.forward / blackCalculator.strike)/blackCalculator.stdDev + 0.5*blackCalculator.stdDev;
blackCalculator.d2 = blackCalculator.d1 - blackCalculator.stdDev;
 normalDistStruct currCumNormDist;
initCumNormDist(currCumNormDist);
 blackCalculator.cum_d1 = cumNormDistOp(currCumNormDist, blackCalculator.d1);
blackCalculator.cum_d2 = cumNormDistOp(currCumNormDist, blackCalculator.d2);
blackCalculator.n_d1 = cumNormDistDeriv(currCumNormDist, blackCalculator.d1);
blackCalculator.n_d2 = cumNormDistDeriv(currCumNormDist, blackCalculator.d2);
 blackCalculator.x = payoff.strike;
blackCalculator.DxDstrike = 1.0;
 blackCalculator.DxDs = 0.0;
 switch (payoff.type) 
{
case CALL: blackCalculator.alpha = blackCalculator.cum_d1;
 blackCalculator.DalphaDd1 = blackCalculator.n_d1;
 blackCalculator.beta = -1.0f*blackCalculator.cum_d2;
 blackCalculator.DbetaDd2 = -1.0f*blackCalculator.n_d2;
 break;
case PUT: blackCalculator.alpha = -1.0+blackCalculator.cum_d1;
 blackCalculator.DalphaDd1 = blackCalculator.n_d1;
 blackCalculator.beta = 1.0-blackCalculator.cum_d2;
 blackCalculator.DbetaDd2 = -1.0f* blackCalculator.n_d2;
 break;
}
}
 __device__ void initBlackCalculator(blackCalcStruct& blackCalc, payoffStruct payoff, float forwardPrice, float stdDev, float riskFreeDiscount)
{
blackCalc.strike = payoff.strike;
blackCalc.forward = forwardPrice;
blackCalc.stdDev = stdDev;
blackCalc.discount = riskFreeDiscount;
blackCalc.variance = stdDev * stdDev;
 initBlackCalcVars(blackCalc, payoff);
}
 __device__ float getResultVal(blackCalcStruct blackCalculator)
{
float result = blackCalculator.discount * (blackCalculator.forward * blackCalculator.alpha + blackCalculator.x * blackCalculator.beta);
return result;
}
 __global__ void getOutValOption(optionInputStruct* options, float* outputVals, int numVals)
{
int optionNum = blockIdx.x * blockDim.x + threadIdx.x;
 if (optionNum < numVals)
{
optionInputStruct threadOption = options[optionNum];
 payoffStruct currPayoff;
currPayoff.type = threadOption.type;
currPayoff.strike = threadOption.strike;
 yieldTermStruct qTS;
qTS.timeYearFraction = threadOption.t;
qTS.forward = threadOption.q;
 yieldTermStruct rTS;
rTS.timeYearFraction = threadOption.t;
rTS.forward = threadOption.r;
 blackVolStruct volTS;
volTS.timeYearFraction = threadOption.t;
volTS.volatility = threadOption.vol;
 blackScholesMertStruct stochProcess;
stochProcess.x0 = threadOption.spot;
stochProcess.dividendTS = qTS;
stochProcess.riskFreeTS = rTS;
stochProcess.blackVolTS = volTS;
 optionStruct currOption;
currOption.payoff = currPayoff;
currOption.yearFractionTime = threadOption.t;
currOption.pricingEngine = stochProcess; 
 float variance = getBlackVolBlackVar(currOption.pricingEngine.blackVolTS);
float dividendDiscount = getDiscountOnDividendYield(currOption.yearFractionTime, currOption.pricingEngine.dividendTS);
float riskFreeDiscount = getDiscountOnRiskFreeRate(currOption.yearFractionTime, currOption.pricingEngine.riskFreeTS);
float spot = currOption.pricingEngine.x0; 
 float forwardPrice = spot * dividendDiscount / riskFreeDiscount;
 blackCalcStruct blackCalc;
 initBlackCalculator(blackCalc, currOption.payoff, forwardPrice, sqrt(variance), riskFreeDiscount);
 float resultVal = getResultVal(blackCalc);
 outputVals[optionNum] = resultVal;
}
}
 #endif 
