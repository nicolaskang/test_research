#include "hip/hip_runtime.h"
//monteCarloEngine.cu
//Scott Grauer-Gray
//May 10, 2012
//Function for running Monte Carlo on the GPU

//needed for the monte carlo GPU kernels
#include "monteCarloKernels.cu"

//needed for the monte carlo CPU kernels
#include "monteCarloKernelsCpu.cu"

//needed for hiprand
#include <hiprand/hiprand_kernel.h>

//needed for the thrust library (can be used for summation on GPU)
/* #include <thrust/transform_reduce.h> */
/* #include <thrust/functional.h> */
/* #include <thrust/device_vector.h> */
/* #include <thrust/host_vector.h> */

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#define RISK_VAL 0.06f
#define DIV_VAL 0.0f
#define VOLT_VAL 0.200f
#define UNDERLYING_VAL 30.0f
#define STRIKE_VAL 40.0f
#define DISCOUNT_VAL 0.94176453358424872f



//initialize the inputs
void initializeInputs(dataType* samplePrices, dataType* sampleWeights, dataType* times)
{
}

//run monte carlo...
void runMonteCarlo()
{
	//int nSamplesArray[] = {100, 200, 500, 1000, 2000, 5000, 10000, 20000, 50000, 100000, 200000, 500000};//, 					1000000, 2000000};//, 5000000};//, 10000000, 20000000};

	int numSamples = 400000;

	//for (int numTime=0; numTime < 12; numTime++)
	{
		//int numSamples = nSamplesArray[numTime];

		printf("Number of Samples: %d\n\n", numSamples);

		//declare and initialize the struct used for the option
		monteCarloOptionStruct optionStruct;
		optionStruct.riskVal = RISK_VAL;
		optionStruct.divVal = DIV_VAL;
		optionStruct.voltVal = VOLT_VAL;
		optionStruct.underlyingVal = UNDERLYING_VAL;
		optionStruct.strikeVal = STRIKE_VAL;
		optionStruct.discountVal = DISCOUNT_VAL;


		//declare pointers for data on CPU
		dataType* samplePrices;
		dataType* sampleWeights;
		dataType* times;
		monteCarloOptionStruct* optionStructs;

		//allocate space for data on CPU
		samplePrices = (dataType*)malloc(NUM_OPTIONS*numSamples*sizeof(dataType));
		sampleWeights = (dataType*)malloc(NUM_OPTIONS*numSamples*sizeof(dataType));
		times = (dataType*)malloc(NUM_OPTIONS*numSamples*sizeof(dataType));
		optionStructs = (monteCarloOptionStruct*)malloc(NUM_OPTIONS*sizeof(monteCarloOptionStruct));

		long seconds, useconds;
		dataType mtimeGpu, mtimeCpu;   
		struct timeval start;
		struct timeval end;

		for (int optNum = 0; optNum < NUM_OPTIONS; optNum++)
		{
			optionStructs[optNum] = optionStruct;
		}

		//initialize values for data on CPU
		//declare pointers for data on GPU
		dataType* samplePricesGpu;
		dataType* sampleWeightsGpu;
		dataType* timesGpu;
		monteCarloOptionStruct* optionStructsGpu;

		//declare what's necessary to use hiprand
		hiprandState* devStates;

		/* Allocate space for prng states on device */
		hipMalloc (( void **) & devStates , numSamples * sizeof ( hiprandState ) );

		//allocate space for data on GPU
		hipMalloc(&samplePricesGpu, NUM_OPTIONS*numSamples*sizeof(dataType));
		hipMalloc(&sampleWeightsGpu, NUM_OPTIONS*numSamples*sizeof(dataType));
		hipMalloc(&timesGpu, NUM_OPTIONS*numSamples*sizeof(dataType));
		hipMalloc(&optionStructsGpu, NUM_OPTIONS*sizeof(monteCarloOptionStruct));

		//transfer data to GPU
		hipMemcpy(samplePricesGpu, samplePrices, NUM_OPTIONS*numSamples*sizeof(dataType), hipMemcpyHostToDevice);
		hipMemcpy(sampleWeightsGpu, sampleWeights, NUM_OPTIONS*numSamples*sizeof(dataType), hipMemcpyHostToDevice);
		hipMemcpy(timesGpu, times, NUM_OPTIONS*numSamples*sizeof(dataType), hipMemcpyHostToDevice);
		hipMemcpy(optionStructsGpu, optionStructs, NUM_OPTIONS*sizeof(monteCarloOptionStruct), hipMemcpyHostToDevice);

	
	
		srand(time(NULL));

		//for (int numTime=0; numTime < 100; numTime++)
		{

		/* initialize random seed: */
		srand ( rand());

		printf("Run on GPU\n");
		gettimeofday(&start, NULL);

		// setup execution parameters
		dim3 grid1( (size_t)ceil((dataType)numSamples / ((dataType)THREAD_BLOCK_SIZE)), 1, 1);
		dim3 threads1( THREAD_BLOCK_SIZE, 1, 1);

		//initializes the states for the random number generator		
		setup_kernel <<< grid1, threads1 >>> (devStates, rand(), numSamples);

		hipDeviceSynchronize();

		// setup execution parameters
	    	dim3 grid2( (size_t)ceil((dataType)numSamples / (THREAD_BLOCK_SIZE)), 1, 1);
	    	dim3 threads2( THREAD_BLOCK_SIZE, 1, 1);

		monteCarloGpuKernel <<< grid2, threads2 >>>(samplePricesGpu, sampleWeightsGpu, timesGpu, (1.0f / (dataType)SEQUENCE_LENGTH), devStates, optionStructsGpu, numSamples);

		hipDeviceSynchronize();

		gettimeofday(&end, NULL);

		seconds  = end.tv_sec  - start.tv_sec;
	    	useconds = end.tv_usec - start.tv_usec;

	    	mtimeGpu = ((seconds) * 1000 + ((dataType)useconds)/1000.0) + 0.5;


	    	printf("Processing time on GPU (CUDA): %f (ms)\n", mtimeGpu);

		// remove comments to perform summation on GPU using thrust library
		/* //create a pointer to point to the prices */
		/* thrust::device_ptr<dataType> devicePointerPrices(samplePricesGpu);	 */

		/* //compute the value for each option */
		/* for (int numOpt=0; numOpt < NUM_OPTIONS; numOpt++) */
		/* { */
		/* 	//use the reduction function in thrust to retrieve the summation of the prices */
		/* 	dataType sum = thrust::reduce(devicePointerPrices + numOpt*numSamples, devicePointerPrices + (numOpt+1)*(numSamples), (dataType) 0, thrust::plus<dataType>()); */

		/* 	dataType avgPrice = sum / numSamples; */
		/* 	printf("avgPrice: %f\n", avgPrice); */
		/* } */

	

		//transfer data back to host
		hipMemcpy(samplePrices, samplePricesGpu, numSamples*sizeof(dataType), hipMemcpyDeviceToHost);
	
		hipMemcpy(sampleWeights, sampleWeightsGpu, numSamples*sizeof(dataType), hipMemcpyDeviceToHost);
		hipMemcpy(times, timesGpu, numSamples*sizeof(dataType), hipMemcpyDeviceToHost);


		//retrieve the average price
		dataType cumPrice = 0.0f;

		//add all the computed prices together
		for (int numSamp = 0; numSamp < numSamples; numSamp++)
		{
			cumPrice += samplePrices[numSamp];
		}

		dataType avgPrice = cumPrice / numSamples;
		printf("Average Price (GPU computation): %f\n\n", avgPrice);
	

		}

		//free memory space on the GPU
		hipFree(samplePricesGpu);
		hipFree(sampleWeightsGpu);
		hipFree(timesGpu);
		hipFree(optionStructsGpu);

		//free memory space on the CPU
		free(samplePrices);
		free(sampleWeights);
		free(times);

		
		//declare pointers for data on CPU
		dataType* samplePricesCpu;
		dataType* sampleWeightsCpu;
		dataType* timesCpu;

		//allocate space for data on CPU
		samplePricesCpu = (dataType*)malloc(numSamples*sizeof(dataType));
		sampleWeightsCpu = (dataType*)malloc(numSamples*sizeof(dataType));
		timesCpu = (dataType*)malloc(numSamples*sizeof(dataType));

	   
		gettimeofday(&start, NULL);
		 
		monteCarloGpuKernelCpu(samplePricesCpu, sampleWeightsCpu, timesCpu, (1.0f / (dataType)SEQUENCE_LENGTH), optionStructs, numSamples);


		gettimeofday(&end, NULL);

		seconds  = end.tv_sec  - start.tv_sec;
		useconds = end.tv_usec - start.tv_usec;

		mtimeCpu = ((seconds) * 1000 + ((dataType)useconds)/1000.0) + 0.5;
		printf("Run on CPU\n");
		printf("Processing time on CPU: %f (ms)\n", mtimeCpu);	

		//retrieve the average price
		dataType cumPrice = 0.0f;

		//add all the computed prices together
		for (int numSamp = 0; numSamp < numSamples; numSamp++)
		{
			cumPrice += samplePricesCpu[numSamp];
		}

		dataType avgPrice = cumPrice / numSamples;
		printf("Average Price (CPU computation): %f\n\n", avgPrice);

		printf("Speedup on GPU: %f\n", mtimeCpu / mtimeGpu);

		//free memory space on the CPU
		free(samplePricesCpu);
		free(sampleWeightsCpu);
		free(timesCpu);
		free(optionStructs);

	}
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
	runMonteCarlo();

	char c;
	c = getchar();
	printf("%c\n", c);
}
